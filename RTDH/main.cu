#include "hip/hip_runtime.h"
//Fix some annoying warnings
#define _CRT_SECURE_NO_DEPRECATE

//GLEW
#define GLEW_STATIC
#include <GL\glew.h>

//GLM
#include <glm/glm.hpp>
#include "glm/gtc/matrix_transform.hpp"

//CUDA
#include "hip/hip_runtime.h"
#include ""

//Project specific includes
#include <cuda_gl_interop.h>//Visualization
#include "hipfft/hipfftXt.h"		//CUDA FFT
#include "hip/hip_runtime_api.h"	//heckCudaErrors

#include "RTDH_utility.h"	
#include "RTDH_GLFW.h"
#include "RTDH_CUDA.h"

//Vimba stuff
#include "ApiController.h"

#include "LoadSaveSettings.h"

//Other
#include <iostream>

//GLFW
#include <GLFW\glfw3.h>

//Pointwise multiplication of two Complex arrays
__global__ void matrixMulComplexPointw(Complex* A, Complex* B, Complex* C, int M, int N);

//Transforms hipfftComplex into a float by calculating the magnitude
__global__ void cufftComplex2MagnitudeF(float *vbo_dptr, Complex *z, const int M, const int N, Complex* d_isNan);

//Checkerboard function that ensures the quadrants are in the right place after fourier-transforming.
__global__ void checkerBoard(Complex* A,const int M,const int N);

//Transforms raw camera input into a normalized float. This will change later on
__global__ void unsignedChar2cufftComplex(unsigned char *A,int M, int N);


//This getting out of hand, just add its contents to main() ffs !
void mainLoop(	GLFWwindow* window, 
				GLuint shaderprogram, 
				GLuint projection_Handle, 
				reconParameters parameters, 
				hipGraphicsResource *cuda_vbo_resource, 
				Complex* d_recorded_hologram, 
				Complex* d_chirp,
				Complex* d_propagated,
				hipfftHandle plan,
				hipfftResult result,
				std::string strCameraID,
				AVT::VmbAPI::Examples::ApiController apiController,
				AVT::VmbAPI::FramePtr pFrame,
				unsigned char* d_recorded_hologram_uchar);

#define PI	3.1415926535897932384626433832795028841971693993751058209749
#define PI2 1.570796326794896619231321691639751442098584699687552910487


int main(){
	//Redirect stderror to log.txt.
	FILE* logfile = freopen("log.txt", "w", stderr);
	printTime(logfile);

	//Initialize the Vimba API and print some info.
	AVT::VmbAPI::Examples::ApiController apiController;

	std::cout << "Vimba Version V " << apiController.GetVersion() << "\n";

	//Start the API
	VmbErrorType vmb_err = VmbErrorSuccess;
	vmb_err = apiController.StartUp();
	if(vmb_err != VmbErrorSuccess){
		fprintf(stderr,"%s: line %d: Vimba API Error: apiController.Startup() failed. \n",__FILE__,__LINE__);
		exit(EXIT_FAILURE); 
	}
	
	//Look for cameras
	std::string strCameraID;
	AVT::VmbAPI::CameraPtr pCamera;
	AVT::VmbAPI::CameraPtrVector cameraList = apiController.GetCameraList();
	if(cameraList.size() == 0){
		fprintf(stderr,"Error: couldn't find a camera. Shutting down... \n");
		apiController.ShutDown();
		exit(EXIT_FAILURE);
	}
	else{
		//If a camera is found, store its pointer.
		pCamera=cameraList[0];
		vmb_err = pCamera->GetID(strCameraID);
		if(vmb_err != VmbErrorSuccess){
			printVimbaError(vmb_err); apiController.ShutDown(); exit(EXIT_FAILURE);}

		//Open the camera and load its settings.
		vmb_err = pCamera->Open(VmbAccessModeFull);
		AVT::VmbAPI::StringVector loadedFeatures;
        AVT::VmbAPI::StringVector missingFeatures;
        vmb_err = AVT::VmbAPI::Examples::LoadSaveSettings::LoadFromFile(pCamera, "CameraSettings.xml", loadedFeatures, missingFeatures, false);
		if(vmb_err != VmbErrorSuccess){
				printVimbaError(vmb_err); apiController.ShutDown(); exit(EXIT_FAILURE);}
		vmb_err = pCamera->Close();
		if(vmb_err != VmbErrorSuccess){
				printVimbaError(vmb_err); apiController.ShutDown(); exit(EXIT_FAILURE);}
	}
	
	//This will be removed later on.
	AVT::VmbAPI::FramePtr pFrame;
	vmb_err = apiController.AcquireSingleImage(strCameraID, pFrame);
	if(vmb_err != VmbErrorSuccess){
		printVimbaError(vmb_err); apiController.ShutDown(); exit(EXIT_FAILURE);}

	
	//=========================INITIALIZATION==========================
	
	//Read the reconstruction parameters. 
	reconParameters parameters;
	read_parameters("parameters.txt", &parameters);

	VmbUint32_t frameWidth = 0;
	VmbUint32_t frameHeight = 0;
	pFrame->GetWidth(frameWidth);
	pFrame->GetHeight(frameHeight);

	//Override the parameters supplied in the file (which will be obsolete anyway); 
	parameters.M=frameHeight;
	parameters.N=frameWidth;

	//Initialize the GLFW window
	//GLFWwindow *window = initGLFW(parameters.N, parameters.M); 
	GLFWwindow *window = initGLFW((int)parameters.N/4, (int) parameters.M/4); 

	//Set a few callbacks
	glfwSetWindowSizeCallback(window, window_size_callback);
	glfwSetKeyCallback(window, key_callback);

	//Search for CUDA devices and pick the best-suited one. 
	findCUDAGLDevices();

	//Allocate and set up the chirp-function, copy it to the GPU memory.
	Complex* h_chirp = (Complex*)malloc(sizeof(Complex)*parameters.N*parameters.M);
	if (h_chirp == NULL){ printError(); exit(EXIT_FAILURE); }

	construct_chirp(h_chirp, parameters.M, parameters.N, parameters.lambda, parameters.rec_dist, parameters.pixel_y, parameters.pixel_x);

	Complex* d_chirp;
	checkCudaErrors(hipMalloc((void**)&d_chirp, sizeof(Complex)*parameters.M*parameters.N));

	checkCudaErrors(hipMemcpy(d_chirp, h_chirp, sizeof(Complex)*parameters.M*parameters.N, hipMemcpyHostToDevice));
	
	//Set up the grid
	dim3 block(16, 16, 1);
	dim3 grid((unsigned int)parameters.M / block.x+1, (unsigned int)parameters.N / block.y+1, 1);

	checkerBoard << <grid, block >> >(d_chirp, parameters.M, parameters.N);
	checkCudaErrors(hipGetLastError());

	//Read the recorded hologram from a file. This will be replaced by the CCD later on.
	Complex* h_recorded_hologram = (Complex*)malloc(sizeof(Complex)*parameters.M*parameters.N);
	if (h_recorded_hologram == NULL){ printError(); exit(EXIT_FAILURE); }



	//The binary must be in single-precision row-major order !!!
	float* h_recorded_hologram_real = read_data("recorded_hologram.bin");

	for (int i = 0; i < parameters.M*parameters.N; i++){
		h_recorded_hologram[i].x =  h_recorded_hologram_real[i];
		h_recorded_hologram[i].y = 0.0;
	}
	
	//Copy the hologram to the GPU
	Complex* d_recorded_hologram;
	checkCudaErrors(hipMalloc((void**)&d_recorded_hologram, sizeof(Complex)*parameters.M*parameters.N));

	checkCudaErrors(hipMemcpy(d_recorded_hologram,h_recorded_hologram,sizeof(Complex)*parameters.M*parameters.N,hipMemcpyHostToDevice));

	unsigned char* d_recorded_hologram_uchar;
	checkCudaErrors(hipMalloc((void**)&d_recorded_hologram_uchar,sizeof(unsigned char)*parameters.M*parameters.N));

	//Copy the recorded image to the device
	VmbUchar_t *pImage;
	vmb_err = pFrame->GetImage(pImage);
	checkCudaErrors(hipMemcpy(	d_recorded_hologram_uchar,pImage,
								sizeof(unsigned char)*parameters.M*parameters.N,
								hipMemcpyHostToDevice));

	Complex* d_propagated;
	checkCudaErrors(hipMalloc((void**)&d_propagated, sizeof(Complex)*parameters.M*parameters.N));


	//We'll use a vertex array object with two VBO's. The first will house the vertex positions, the second will 
	//house the magnitude that will be calculated with a kernel. 
	
	GLuint vao;
	GLuint vbo[2];
	
	//Create the vertex array object and two vertex buffer object names.
	glGenVertexArrays(1, &vao);
	checkGLError(glGetError());
	
	glBindVertexArray(vao);
	checkGLError(glGetError());

	glGenBuffers(2, vbo);
	checkGLError(glGetError());

	//First let's set up all vertices in the first vbo. 
	glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);
	checkGLError(glGetError());

	
	//Calculate the position of each vertex (one for every pixel in the image). 
	float u, v, x, y;
	int k = 0;

	
	float *vertices = (float *) malloc(parameters.M*parameters.N * 2 * sizeof(float));
	
	for (int i = 0; i < parameters.M; i++){
		for (int j = 0; j < parameters.N; j++){
			u = (float)j - 0.5f*(float)parameters.N;
			v = (float)i - 0.5f*(float)parameters.M;
			x = (u) / (0.5f*(float)parameters.N);
			y = -(v) / (0.5f*(float)parameters.M);

			vertices[k] = x;
			vertices[k + 1] = y;
			k += 2;
		}
	}
	
	//Load these vertex coordinates into the first vbo
	glBufferData(GL_ARRAY_BUFFER, parameters.N*parameters.M * 2 * sizeof(GLfloat), vertices, GL_DYNAMIC_DRAW);
	checkGLError(glGetError());

	glVertexAttribPointer(0,2,GL_FLOAT,GL_FALSE,0,0);
	checkGLError(glGetError());

	glEnableVertexAttribArray(0);
	checkGLError(glGetError());

	//Bind the second VBO that will contain the magnitude of each complex number. 
	glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);
	checkGLError(glGetError());

	glVertexAttribPointer(1, 1, GL_FLOAT, GL_FALSE, 0, 0);
	checkGLError(glGetError());

	glEnableVertexAttribArray(1);
	checkGLError(glGetError());

	//IDEE: schrijf een kernel van cufftcomplex - > VBO
	//This doesn't work, h_recorded_hologram is an array of structs with x- and y- fields, and glBufferData expects an array of the form 
	// x0 y0 | x1 y1 | ... | xn yn

	//This is the VBO that the complex magnitudes will be written to for visualization.
	glBufferData(GL_ARRAY_BUFFER, parameters.M*parameters.N * 1 * sizeof(GLfloat), 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	checkGLError(glGetError());

	hipGraphicsResource *cuda_vbo_resource;
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo[1], cudaGraphicsMapFlagsWriteDiscard));

	//Compile vertex and fragment shaders

	GLuint shaderprogram = initShaders();
	checkGLError(glGetError());

	// Set up cuFFT stuff
	hipfftComplex* d_reconstructed;
	hipMalloc((void**)&d_reconstructed, sizeof(hipfftComplex)*parameters.M*parameters.N);

	//Set up plan
	hipfftResult result = HIPFFT_SUCCESS;
	hipfftHandle plan;
	result = hipfftPlan2d(&plan, parameters.M, parameters.N, HIPFFT_C2C);
	if (result != HIPFFT_SUCCESS) { printCufftError(); exit(EXIT_FAILURE); }
	
	//=========================MAIN LOOP==========================

	GLuint projection_Handle= glGetUniformLocation(shaderprogram, "Projection");

	mainLoop(	window, 
				shaderprogram, 
				projection_Handle, 
				parameters, 
				cuda_vbo_resource, 
				d_recorded_hologram, 
				d_chirp,
				d_propagated, 
				plan,
				result,
				strCameraID,
				apiController,
				pFrame,
				d_recorded_hologram_uchar);

	//Export the last reconstructed frame. 
	Complex* h_reconstructed=(Complex*) malloc(sizeof(Complex)*parameters.M*parameters.N);
	checkCudaErrors(hipMemcpy(h_reconstructed, d_propagated, sizeof(Complex)*parameters.M*parameters.N, hipMemcpyDeviceToHost));

	export_complex_data("reconstructed_hologram.bin", h_reconstructed, parameters.M*parameters.N);
	

	//Cleanup

	
	checkCudaErrors(hipFree(d_recorded_hologram));
	checkCudaErrors(hipFree(d_recorded_hologram_uchar));
	checkCudaErrors(hipFree(d_chirp));
	checkCudaErrors(hipFree(d_propagated));

	

	free(vertices);
	free(h_recorded_hologram);
	free(h_chirp);
	free(h_reconstructed);
	
	glfwTerminate();

	//Gracefully (lol, as if) end Vimba stuff.
	if(pCamera != NULL){pCamera->Close();}
	apiController.ShutDown();

	
	fprintf(stderr, "No errors (that I'm aware of)! \n");
	fclose(logfile);

	return 0;
};

//Not needed 
__global__ void restrictToRange(float* vbo_magnitude, const int M, const int N){
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < M && j < N){
		float min_Val = 5.7*7000.0;
		float max_Val = 1.11*70000.0;
		vbo_magnitude[i*N + j] = (vbo_magnitude[i*N + j] - min_Val) / (max_Val - min_Val); //This is a constant so we might want to calculate this beforehand. 
	}
};

__global__ void cufftComplex2MagnitudeF(float* vbo_mapped_pointer, Complex *z, const int M, const int N){
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < M && j < N){
	float magnitude = sqrt(pow(z[i*N + j].x, (float)2) + pow(z[i*N + j].y, (float)2));
	vbo_mapped_pointer[i*N + j] = magnitude;// log(1.0 + magnitude);// / sqrt((float)M*(float)N)) / 75.0; //This is a constant so we might want to calculate this beforehand. 
	}
};

__global__ void matrixMulComplexPointw(Complex* A, Complex* B, Complex* C, int M, int N){
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < M && j < N){
		C[i*N + j].x = A[i*N + j].x*B[i*N + j].x;
		C[i*N + j].y = A[i*N + j].y*B[i*N + j].y;		
	}
}

__global__ void checkerBoard(Complex* A, int M, int N){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < M && j < N){
		A[i*N + j].x = A[i*N + j].x*(float)((i + j) % 2) -A[i*N + j].x*(float)(1 - ((i + j) % 2));
		A[i*N + j].y = A[i*N + j].y*(float)((i + j) % 2) -A[i*N + j].y*(float)(1 - ((i + j) % 2));
	}
}

__global__ void unsignedChar2cufftComplex(Complex* z, unsigned char *A, int M, int N){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	if (i < M && j < N){
		z[i*N+j].x=(float) A[i*N+j]/255.0;
		z[i*N+j].y=0.0;
	}
};



void mainLoop(	GLFWwindow* window, 
				GLuint shaderprogram, 
				GLuint projection_Handle, 
				reconParameters parameters, 
				hipGraphicsResource *cuda_vbo_resource, 
				Complex* d_recorded_hologram, 
				Complex* d_chirp,
				Complex* d_propagated,
				hipfftHandle plan,
				hipfftResult result,
				std::string strCameraID,
				AVT::VmbAPI::Examples::ApiController apiController,
				AVT::VmbAPI::FramePtr pFrame,
				unsigned char* d_recorded_hologram_uchar){

	// Measure frametime
	double frameTime = 0.0;
	int fps = 1;
	int fps_prev = 1;
	int framecounter = 1;
	std::string wtitle;
	VmbErrorType vmb_err;
	VmbUchar_t *pImage;

	while (!glfwWindowShouldClose(window))
	{
		//Start measuring frame time
		glfwSetTime(0.0);
		float ratio = (float)parameters.N / (float)parameters.M;
		
		//Set up the grid
		dim3 block(16, 16, 1);
		//I added the +1 because it might round down which can mean that not all pixels are processed in each kernel. 
		dim3 grid((unsigned int)parameters.M / block.x+1, (unsigned int)parameters.N / block.y+1, 1);
		
		//Fetch an image, copy it to the device and convert it
 
		vmb_err = apiController.AcquireSingleImage(strCameraID, pFrame);
		if(vmb_err != VmbErrorSuccess){
		printVimbaError(vmb_err); apiController.ShutDown(); exit(EXIT_FAILURE);}

		
		vmb_err = pFrame->GetImage(pImage);
		checkCudaErrors(hipMemcpy(	d_recorded_hologram_uchar,pImage,
								sizeof(unsigned char)*parameters.M*parameters.N,
								hipMemcpyHostToDevice));

		unsignedChar2cufftComplex<< <grid, block >> >(d_recorded_hologram,d_recorded_hologram_uchar,parameters.M,parameters.N);

		matrixMulComplexPointw << <grid, block >> >(d_chirp, d_recorded_hologram, d_propagated, parameters.M, parameters.N);
		checkCudaErrors(hipGetLastError());

		result = hipfftExecC2C(plan,d_propagated, d_propagated, HIPFFT_FORWARD);
		if (result != HIPFFT_SUCCESS) { printCufftError(); exit(EXIT_FAILURE); }
		
		float *vbo_mapped_pointer; //This is the pointer that we'll write the result to for display in OpenGL.
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));

		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&vbo_mapped_pointer, &num_bytes, cuda_vbo_resource));


		cufftComplex2MagnitudeF << <grid, block >> >(vbo_mapped_pointer, d_recorded_hologram, parameters.M, parameters.N);

		//restrictToRange << <grid, block >> >(dptr, parameters.M, parameters.N);

		checkCudaErrors(hipGetLastError());
		
		// unmap buffer object
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
		
		int w_width, w_height;
		glfwGetWindowSize(window, &w_width, &w_height);
		
		//glm::mat4 Projection = glm::ortho(-(float)parameters.N / (float)w_width, (float)parameters.N / (float)w_width, -(float)parameters.M / (float)w_height, (float)parameters.M / (float)w_height);
		glm::mat4 Projection = glm::ortho(-1.0,1.0,-1.0,1.0);

		glUniformMatrix4fv(projection_Handle, 1, GL_FALSE, &Projection[0][0]);


		glDrawArrays(GL_POINTS, 0, parameters.N*parameters.M);

		glfwSwapBuffers(window);
		glfwPollEvents();

		//Display fps in cmd, not sure if averaging is that useful
		frameTime = glfwGetTime();
		fps_prev = fps;
		fps = (int)(0.5*(1. / frameTime + (float)fps_prev));

		//Update FPS every 15 frames
		framecounter += 1;
		if (framecounter == 15){
			framecounter = 1;
			fprintf(stdout, "\r Frames Per Second: %i             ", fps);
			wtitle = std::to_string(fps);
			glfwSetWindowTitle(window, wtitle.c_str());
		}
	}
};