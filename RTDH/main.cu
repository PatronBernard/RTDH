#include "hip/hip_runtime.h"
//Fix some annoying warnings
#define _CRT_SECURE_NO_DEPRECATE

//GLEW
#define GLEW_STATIC
#include <GL\glew.h>

//GLFW
#include <GLFW\glfw3.h>

//CUDA
#include "hip/hip_runtime.h"
#include ""

//Project specific includes
#include <cuda_gl_interop.h>//Visualization
#include "hipfft/hipfftXt.h"		//CUDA FFT
#include "hip/hip_runtime_api.h"	//heckCudaErrors

#include "RTDH_utility.h"	
#include "RTDH_GLFW.h"
#include "RTDH_CUDA.h"

__global__ void simple_vbo_kernel(float4 *pos, const int width, const int height);

__global__ void cufftComplex2Float(float *vbo_dptr, Complex *z, const int width, const int height);

void mainLoop(GLFWwindow* window, reconParameters parameters, hipGraphicsResource *cuda_vbo_resource, Complex* d_recorded_hologram);


//TODO: -fix everything, reorganize headers so it makes sense
//		-perhaps generate a test hologram at a smaller size?
int main(){

	//=========================INITIALIZATION==========================
	//Redirect stderror to log.txt.
	FILE* logfile = freopen("log.txt", "w", stderr);
	printTime(logfile);

	//Read the reconstruction parameters. 
	reconParameters parameters;
	read_parameters("parameters.txt", &parameters);

	//Initialize the GLFW window
	GLFWwindow *window = initGLFW(parameters.N, parameters.M); 

	//Set a few callbacks
	glfwSetWindowSizeCallback(window, window_size_callback);
	glfwSetKeyCallback(window, key_callback);

	//Search for CUDA devices and pick the best-suited one. 
	findCUDAGLDevices();

	//Read the recorded hologram from a file. This will be replaced by the CCD later on.
	Complex* h_recorded_hologram = (Complex*)malloc(sizeof(Complex)*parameters.N*parameters.M);
	if (h_recorded_hologram == NULL){ printError(); exit(EXIT_FAILURE); }
	float* h_recorded_hologram_real = read_data("recorded_hologram_scaled.bin");

	for (int i = 0; i < parameters.M*parameters.N; i++){
		h_recorded_hologram[i].x =  h_recorded_hologram_real[i];
		h_recorded_hologram[i].y = 0.0;
	}
	
	//Copy the hologram to the GPU
	Complex* d_recorded_hologram;
	checkCudaErrors(hipMalloc((void**)&d_recorded_hologram, sizeof(Complex)*parameters.N*parameters.M));

	checkCudaErrors(hipMemcpy(d_recorded_hologram,h_recorded_hologram,sizeof(Complex)*parameters.N*parameters.M,hipMemcpyHostToDevice));

	//We'll use a vertex array object with two VBO's. The first will house the vertex positions, the second will 
	//house their colours/complex value. We cannot put the positions and complex values in a single VBO because cuFFT requires
	//a float2. 
	
	GLuint vao;
	GLuint vbo[2];
	
	//Create the vertex array object and two vertex buffer object names.
	glGenVertexArrays(1, &vao);
	checkGLError(glGetError());
	
	glBindVertexArray(vao);
	checkGLError(glGetError());

	glGenBuffers(2, vbo);
	checkGLError(glGetError());

	glBindBuffer(GL_ARRAY_BUFFER, vbo[0]);
	checkGLError(glGetError());

	
	//Calculate the position of each vertex (one for every pixel in the image). 
	float u, v, x, y;
	int k = 0;

	
	float *position = (float *) malloc(parameters.N*parameters.M * 2 * sizeof(float));
	for (int i = 0; i < parameters.N; i++){
		for (int j = 0; j < parameters.M; j++){
			u = (float)i - 0.5f*(float)parameters.N;
			v = (float)j - 0.5f*(float)parameters.M;
			x = (u) / (0.5f*(float)parameters.N);
			y = (v) / (0.5f*(float)parameters.M);

			position[k] = x;
			position[k + 1] = y;
			k += 2;
		}
	}
	
	//Load these vertex coordinates into the first vbo
	glBufferData(GL_ARRAY_BUFFER, parameters.N*parameters.M * 2 * sizeof(GLfloat), position, GL_DYNAMIC_DRAW);
	checkGLError(glGetError());

	glVertexAttribPointer(0,2,GL_FLOAT,GL_FALSE,0,0);
	checkGLError(glGetError());

	glEnableVertexAttribArray(0);
	checkGLError(glGetError());

	//Bind the second VBO that will contain the magnitude of each complex number. 
	glBindBuffer(GL_ARRAY_BUFFER, vbo[1]);
	checkGLError(glGetError());

	glVertexAttribPointer(0, 1, GL_FLOAT, GL_FALSE, 0, 0);
	checkGLError(glGetError());

	glEnableVertexAttribArray(0);
	checkGLError(glGetError());

	//IDEE: schrijf een kernel van cufftcomplex - > VBO
	//This doesn't work, h_recorded_hologram is an array of structs with x- and y- fields, and glBufferData expects an array of the form 
	// x0 y0 | x1 y1 | ... | xn yn

	//This is the VBO that the complex magnitudes will be written to for visualization.
	glBufferData(GL_ARRAY_BUFFER, parameters.N*parameters.M * 1 * sizeof(GLfloat), 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	checkGLError(glGetError());

	hipGraphicsResource *cuda_vbo_resource;
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo[1], cudaGraphicsMapFlagsWriteDiscard));

	//Compile vertex and fragment shaders

	initShaders();
	checkGLError(glGetError());

	
	//=========================MAIN LOOP==========================

	//checkCudaErrors(hipDeviceReset());

	mainLoop(window, parameters, cuda_vbo_resource, d_recorded_hologram);
	
	glfwTerminate();

	free(position);
	free(h_recorded_hologram);
	checkCudaErrors(hipFree(d_recorded_hologram));

	checkCudaErrors(hipDeviceReset());

	fprintf(stderr, "No errors (that I'm aware of)! \n");
	fclose(logfile);

	return 0;
};

__global__ void simple_vbo_kernel(float4 *pos, const int width, const int height){
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = (float)x / (float)width;
	float v = (float)y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	float w = 0.5f*sqrt(pow(u, 2.0f) + pow(v, 2.0f));

	// write output vertex
	pos[y*width + x] = make_float4(u, v, u, v);
}

__global__ void cufftComplex2Float(float* vbo_magnitude, Complex *z, const int width, const int height){
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;
	//float magnitude = pow(z[j*width + i].x, (float)2) + pow(z[j*width + i].y, (float)2);
	vbo_magnitude[j*width + i] = 1.0; // make_float1(sqrt(magnitude));
};

void mainLoop(GLFWwindow* window, reconParameters parameters, hipGraphicsResource *cuda_vbo_resource, Complex* d_recorded_hologram){
	// Measure frametime
	double frameTime = 0.0;
	int fps = 1;
	int fps_prev = 1;
	int framecounter = 1;
	std::string wtitle;

	while (!glfwWindowShouldClose(window))
	{
		glfwSetTime(0.0);
		//float ratio = (float)parameters.N / (float)parameters.M;
		// handle events

		//Calculate position with CUDA
		// map OpenGL buffer object for writing from CUDA
		
		float *dptr; //This will become a float2 as to be compatible with cuFFT
		checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource, 0));
		
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
			cuda_vbo_resource));
		
		//Run kernel, this will become a simple kernel and hipfftExecC2C call
		//First thing to do: get an external stream to 
		dim3 block(8, 8, 1);
		dim3 grid((unsigned int)parameters.N / block.x, (unsigned int)parameters.M / block.y, 1);
		cufftComplex2Float<<<grid, block >>>(dptr, d_recorded_hologram, parameters.N, parameters.M);
		//checkCudaErrors(hipGetLastError());
		
		// unmap buffer object
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
		

		glDrawArrays(GL_POINTS, 0, parameters.N*parameters.M);

		glfwSwapBuffers(window);
		glfwPollEvents();

		//Display fps in cmd, not sure if averaging is that useful
		frameTime = glfwGetTime();
		fps_prev = fps;
		fps = (int)(0.5*(1. / frameTime + (float)fps_prev));

		//Update FPS every 15 frames
		framecounter += 1;
		if (framecounter == 15){
			framecounter = 1;
			fprintf(stdout, "\r Frames Per Second: %i             ", fps);
			wtitle = std::to_string(fps);
			glfwSetWindowTitle(window, wtitle.c_str());
		}
	}
};